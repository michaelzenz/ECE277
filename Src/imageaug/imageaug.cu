#include <stdio.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <vector>


class Conv{

	private:
	static byte* d_data;
	static int capa;

	Conv(){}
	Conv(Conv const&);
	void operator =(Conv const&);


	public:

	Conv(Conv const&)            = delete;
	void operator=(Conv const&)  = delete;

	static Conv getInstance(){
		capa=0;
		static Conv instance;

		return instance;
	}

	static void Convolution(byte* data, int kernel, int chan){
	
	}
};

int main()
{
    int *a, *b, *c, *golden;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );
	

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );
	golden = (int *)malloc(size);

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		golden[i] = a[i] + b[i];
		c[i] = 0;
	}


	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	vector_add <<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	bool pass = true;
	for (int i = 0; i < N; i++) {
		if (golden[i] != c[i])
			pass = false;
	}
	
	if (pass)
		printf("PASS\n");
	else
		printf("FAIL\n");

	printf("print your name and id\n");

	free(a);
	free(b);
	free(c);
	free(golden);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} 
